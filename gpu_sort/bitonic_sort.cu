#include "hip/hip_runtime.h"
#include "bitonic_sort.h"

__global__ void bitonic_sort_step(int *ptr, int j, int k) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;
    int temp;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            if (ptr[ixj] < ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        } else {
            if (ptr[ixj] > ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(int *start, int *end, unsigned long long real_bytes) {
    int *kernel_ptr;
    size_t size = (end - start);
    size_t bytes = size * sizeof(int);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);

    hipMalloc((void **) &kernel_ptr, bytes);
    hipMemcpy(kernel_ptr, start, bytes, hipMemcpyHostToDevice);

    dim3 blocks(std::max(1ull, size / props.maxThreadsPerBlock), 1);
    dim3 threads(props.maxThreadsPerBlock, 1);

    int j, k;
    for (k = 2; k <= size; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step<<<blocks, threads>>>(kernel_ptr, j, k);
        }
    }
    hipMemcpy(start, kernel_ptr, real_bytes, hipMemcpyDeviceToHost);
    hipFree(kernel_ptr);
}

/*
void cpu_gpu_sort(int * start, int * end, unsigned int core_count) {
    using ull = unsigned long long;
    ull cpu_size = (end - start) / 5 * 2;
    ull gpu_size = (end - start) / 5 * 3 + ((end - start) % 5);
    ull cpu_bytes = cpu_size * sizeof(int);
    ull gpu_bytes = gpu_size * sizeof(int);
    int *arr1 = new int[cpu_size];
    int *arr2;

    ull padded_size = 1;
    while (padded_size < gpu_size) {
        padded_size <<= 1;
    }
    arr2 = new int[padded_size];

    memcpy(arr1, start, cpu_bytes);
    memcpy(arr2, start + cpu_size, gpu_bytes);
    for (ull i = gpu_size; i < padded_size; i++) {
        arr2[i] = INT_MAX;
    }

    auto t = std::thread(bitonic_sort, arr2, arr2 + padded_size, gpu_bytes);
    par_sort(arr1, arr1 + cpu_size, cpu_size, cpu_size / core_count, std::less<>{});
    t.join();

    std::merge(arr1, arr1 + cpu_size, arr2, arr2 + gpu_size, start);

    delete[] arr1;
    delete[] arr2;
}
*/