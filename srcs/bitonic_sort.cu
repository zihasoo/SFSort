
#include <hip/hip_runtime.h>
#include <utility>
#include <algorithm>

#define THREADS 512 // 2^9
#define BLOCKS 32768 // 2^15
#define NUM_VALS THREADS*BLOCKS

template<typename iter, typename comp>
__global__ void bitonic_sort_step(iter kernel_iter, int j, int k, comp cmp) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            if (cmp(kernel_iter[ixj], kernel_iter[i])) {
                std::iter_swap(kernel_iter + i, kernel_iter + ixj);
            }
        }
        else {
            if (cmp(kernel_iter[i], kernel_iter[ixj])) {
                std::iter_swap(kernel_iter + i, kernel_iter + ixj);
            }
        }
    }
}

template<typename iter, typename comp>
void bitonic_sort(iter begin, iter end, comp cmp) {
    iter kernel_iter;
    size_t size = (end - begin) * sizeof(decltype(*begin));

    hipMalloc(static_cast<void **>(&kernel_iter), size);
    hipMemcpy(kernel_iter, begin, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    int j, k;
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step<<<blocks, threads>>>(kernel_iter, j, k, cmp);
        }
    }
    hipMemcpy(begin, kernel_iter, size, hipMemcpyDeviceToHost);
    hipFree(kernel_iter);
}
