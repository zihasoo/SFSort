#include "hip/hip_runtime.h"
#include <utility>
#include "bitonic_sort.h"

__global__ void bitonic_sort_step(int* ptr, int j, int k) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;
    int temp;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            if (ptr[ixj] < ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        }
        else {
            if (ptr[ixj] > ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(int* begin, int* end) {
    int* kernel_ptr;
    size_t size = (end - begin) * sizeof(int);

    hipMalloc((void **)(&kernel_ptr), size);
    hipMemcpy(kernel_ptr, begin, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    int j, k;
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step<<<blocks, threads>>>(kernel_ptr, j, k);
        }
    }
    hipMemcpy(begin, kernel_ptr, size, hipMemcpyDeviceToHost);
    hipFree(kernel_ptr);
}
