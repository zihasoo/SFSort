#include "hip/hip_runtime.h"
#include "bitonic_sort.h"

__global__ void bitonic_sort_step(int *ptr, int j, int k) {
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;
    int temp;

    if ((ixj) > i) {
        if ((i & k) == 0) {
            if (ptr[ixj] < ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        } else {
            if (ptr[ixj] > ptr[i]) {
                temp = ptr[i];
                ptr[i] = ptr[ixj];
                ptr[ixj] = temp;
            }
        }
    }
}

void checkCuda(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << '\n';
    }
}

void bitonic_sort(int *begin, int *end, unsigned long long real_bytes) {
    int *kernel_ptr;
    size_t size = (end - begin);
    size_t bytes = size * sizeof(int);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);

    checkCuda(hipMalloc((void **) &kernel_ptr, bytes));
    hipMemcpy(kernel_ptr, begin, bytes, hipMemcpyHostToDevice);

    dim3 blocks(std::max(1ull, size / props.maxThreadsPerBlock), 1);
    dim3 threads(props.maxThreadsPerBlock, 1);

    int j, k;
    for (k = 2; k <= size; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step<<<blocks, threads>>>(kernel_ptr, j, k);
        }
    }
    checkCuda(hipMemcpy(begin, kernel_ptr, real_bytes, hipMemcpyDeviceToHost));
    checkCuda(hipFree(kernel_ptr));
}
